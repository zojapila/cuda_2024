
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

#define cudaCheckError()                                                                     \
    {                                                                                        \
        hipError_t e = hipGetLastError();                                                  \
        if (e != hipSuccess)                                                                \
        {                                                                                    \
            printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
            exit(EXIT_FAILURE);                                                              \
        }                                                                                    \
    }

#define IDX(row, col, LDA) ((row) * (LDA) + (col))

__global__ void add_v1(int *a, int *b, int *c, int N, int M)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < N && j < M)
    {
        int idx = IDX(i, j, M);
        c[idx] = a[idx] + b[idx];
    }
}
__global__ void add_v2(int *a, int *b, int *c, int N, int M)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N && j < M)
    {
        int idx = IDX(i, j, M);
        c[idx] = a[idx] + b[idx];
    }
}

int main()
{
    int N = 2 * 1024;
    int M = 2 * 1024;
    int *a, *b, *c;
    dim3 threads(32, 32);
    dim3 blocks(N / threads.x, M / threads.y);

    hipMallocManaged(&a, N * M * sizeof(int));
    hipMallocManaged(&b, N * M * sizeof(int));
    hipMallocManaged(&c, N * M * sizeof(int));

    add_v1<<<blocks, threads>>>(a, b, c, N, M);
    add_v2<<<blocks, threads>>>(a, b, c, N, M);

    hipFree(a);
    hipFree(b);
    hipFree(c);

    hipDeviceSynchronize();
    cudaCheckError();
    return 0;
}
