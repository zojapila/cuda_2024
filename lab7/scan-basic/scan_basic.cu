#include "hip/hip_runtime.h"
#include "scanFiles.h"

#include <stdio.h>

#define SECTION_SIZE 512

void scanSequential(float *output, float *input, int width)
{
    float accumulator = input[0];
    output[0] = accumulator;
    for (int i = 1; i < width; ++i)
    {
        accumulator += input[i];
        output[i] = accumulator;
    }
}

__global__ void scanKernel(float *input, float *output, int width)
{
    //@@ INSERT CODE HERE
    __shared__ float sharedData[SECTION_SIZE];

    int tid = threadIdx.x;
    if (tid < width)
    {
        sharedData[tid] = input[tid];
    }
    __syncthreads();
   for (int i=1; i < width; i *= 2) 
   {
        float temp = 0;
        if (tid >= i) 
        {
            temp = sharedData[tid - i];
        }
        __syncthreads();
        sharedData[tid] += temp;
        __syncthreads();
   }

    if (tid < width)
    {
        output[tid] = sharedData[tid];
    }
}

void launchScanKernel(float *h_output, float *h_input, int width)
{
    //@@ INSERT CODE HERE
    size_t size = width * sizeof(float);

    float *d_input, *d_output;
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    scanKernel<<<1, SECTION_SIZE>>>(d_input, d_output, width);

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

int main(int argc, char *argv[])
{

    // check if number of input args is correct: input and output image filename
    if (argc != 2)
    {
        printf("Wrong number of arguments: exactly 1 arguments needed (input .txt filename)\n");
        return 1;
    }

    // read data
    int inputSize;
    float *inputData = NULL;
    int status = readData(argv[1], &inputData, &inputSize);
    if (status == NO_FILE)
    {
        fprintf(stderr, "%s: No such file or directory.\n", argv[1]);
        return 2;
    }
    else if (status == NO_MEMO)
    {
        fprintf(stderr, "Cannot allocate memory for the input data.\n");
        return 3;
    }

    // reference output
    float *outputRef = (float *)malloc(inputSize * sizeof(float));
    scanSequential(outputRef, inputData, inputSize);

    // launch kernel
    float *outputScan = (float *)malloc(inputSize * sizeof(float));
    launchScanKernel(outputScan, inputData, inputSize);

    // check results
    int nErr = 0;
    for (int i = 0; i < inputSize; ++i)
    {
        if (outputRef[i] != outputScan[i])
        {
            nErr++;
            printf("Error at [%d]: %f seq vs %f par\n", i, outputRef[i], outputScan[i]);
        }
    }
    if (nErr == 0)
    {
        printf("Scan Kernel OK!\n");
    }
    else
    {
        printf("Scan Kernel FAIL! %d/%d errors detected.\n", nErr, inputSize);
    }

    // write output data
    writeData("outSequential.txt", outputRef, inputSize);
    writeData("outParallel.txt", outputScan, inputSize);

    // clean
    free(inputData);
    free(outputRef);
    free(outputScan);

    return 0;
}